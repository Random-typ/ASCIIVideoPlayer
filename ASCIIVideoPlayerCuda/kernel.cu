#include "hip/hip_runtime.h"
﻿//#include "cudaFunctions.h"

#include "VideoPlayer.h"
#include "frameCompute.h"


namespace CudaFrameCompute {
    __global__ void computeRow(uchar* _frame, char* _data, int* _width, size_t* _p0, char* _shadesDevice, int* _shadesDeviceLen)
    {
        int i = threadIdx.x;
        for (int j = 0; j < *_width; j++)
        {
            int offset = *_width * 3 * i + j * 3;

            // calculate grayscale
            unsigned char color = (_frame[offset] + _frame[offset + 1] + _frame[offset + 2]) / 3;
            // find fitting ASCII character
            if (color == 255)
            {
                color = _shadesDevice[*_shadesDeviceLen - 1];
            }
            else {
                color = _shadesDevice[(int)(color / (255.0 / *_shadesDeviceLen))];
            }
            // set ASCII character in final frame
            _data[((*_width) * 2 + 1) * i + (j + 1) * 2] = color;
            _data[((*_width) * 2 + 1) * i + (j + 1) * 2 - 1] = color;
        }
    }

    bool computeFrameAlloc(int _width, const cv::Mat& _frame, const std::string& _ASCIIFrame)
    {
        const char* shades = ".'`^\",:;Il!i><~+_-?][}{1)(|\\/tfjrxnuvczXYUJCLQ0OZmwqpdbkhao*#MW&8%B@";
        int shadesLen = strlen(shades);
        hipError_t cudaErr = hipSetDevice(0);
        if (cudaErr != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
            return false;
        }

        // alloc constant space
        hipMalloc(&CudaASCIIFrame, _ASCIIFrame.size());
        hipMalloc(&frame, _frame.dataend - _frame.datastart);
        
        // alloc constants
        hipMalloc(&width, sizeof(int));
        hipMalloc(&p0, sizeof(size_t));
        hipMalloc(&shadesDevice, strlen(shades));
        hipMalloc(&shadesDeviceLen, sizeof(int));
        
        // set constants these need to be copied once
        hipMemcpy(width, &_width, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(p0, &_frame.size.p[0], sizeof(size_t), hipMemcpyHostToDevice);
        hipMemcpy(shadesDevice, shades, strlen(shades), hipMemcpyHostToDevice);
        hipMemcpy(shadesDeviceLen, &shadesLen, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(CudaASCIIFrame, _ASCIIFrame.data(), _ASCIIFrame.size(), hipMemcpyHostToDevice);

        return true;
    }

    bool computeFrameFree()
    {
        // free allocated 
        hipFree(width);
        hipFree(p0);
        hipFree(CudaASCIIFrame);
        hipFree(frame);
        hipFree(shadesDevice);
        hipFree(shadesDeviceLen);

        hipError_t cudeErr = hipDeviceReset();
        if (cudeErr != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
            return false;
        }
        return true;
    }


    bool computeFrameCuda(cv::Mat& _frame, std::string& _ASCIIFrame, int _width, int _height)
    {
        hipError_t cudaErr;
        // copy image data
        hipMemcpy(frame, _frame.data, _frame.dataend - _frame.datastart, hipMemcpyHostToDevice);

        // launch kernel
        computeRow<<<1, _height>>>(frame, CudaASCIIFrame, width, p0, shadesDevice, shadesDeviceLen);

        cudaErr = hipGetLastError();
        if (cudaErr != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaErr));
            return false;
        }

        // wait for kernel to finish
        cudaErr = hipDeviceSynchronize();
        if (cudaErr != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaErr);
            return false;
        }

        hipMemcpy((void*)_ASCIIFrame.data(), CudaASCIIFrame, _ASCIIFrame.size(), hipMemcpyDeviceToHost);
        return true;
    }
}


void playVideo(std::string _filePath, bool _cudaAccelEnabled)
{
    cv::Mat frame;
    cv::Mat frameResized;

    cv::VideoCapture vidCap;
    // TODO: handle audio

    //vidCap.open(1); // CAM
    vidCap.open(_filePath, cv::CAP_FFMPEG);

    if (!vidCap.isOpened())
    {
        // todo: error msg
        return;
    }

    std::string textFrame;

    int height = 500;
    int width = vidCap.get(cv::CAP_PROP_FRAME_WIDTH) / vidCap.get(cv::CAP_PROP_FRAME_HEIGHT) * height;

    std::vector<int64_t> frameConvertTimes;
    //frameConvertTimes.reserve(vidCap.get(cv::CAP_PROP_FRAME_COUNT));

    textFrame.resize(height * (width * 2 + 1));
    // add new line to end of each row
    for (size_t i = 1; i < height; i++)
    {
        textFrame[(width * 2 + 1) * i] = '\n';
    }

    // measuring execute time
    long double computeFramesTotal = 0;
    long double printFramesTotal = 0;

    vidCap.get(cv::CAP_PROP_FRAME_COUNT);
    int frameTime = 1000 / vidCap.get(cv::CAP_PROP_FPS) * 1000;// frame time in MicroS
    bool cudeAllocated = false;
    for (auto startTime = std::chrono::high_resolution_clock::now();; startTime = std::chrono::high_resolution_clock::now()) {
        // read frame
        vidCap.read(frame);

        if (frame.empty())
            break;// video has ended

        // TODO: skip frames if to slow
                // resize frame
        cv::resize(frame, frameResized, cv::Size(width, height));// TODO: find best resize method

        // init cuda
        if (_cudaAccelEnabled && !cudeAllocated)
        {
#ifndef __CUDA_RUNTIME_H__
            std::cout << "No Cuda runtime detected!\n";
            return;
#else 
            if (!CudaFrameCompute::computeFrameAlloc(width, frameResized, textFrame))
            {
            	return;
            }
            cudeAllocated = true;
#endif
        }


        // measure execute time
        auto computeFrameStart = std::chrono::high_resolution_clock::now();
        // create ASCII frame
        if (_cudaAccelEnabled)
        {// GPU
#ifdef __CUDA_RUNTIME_H__
            CudaFrameCompute::computeFrameCuda(frameResized, textFrame, width, height);
#endif // !__CUDA_RUNTIME_H__
        }
        else
        {// CPU
            VideoPlayer::computeFrame(frameResized, width, textFrame);
        }
        computeFramesTotal = (std::chrono::high_resolution_clock::now() - computeFrameStart).count() / 1000;
        std::string info = "Compute Time: " + std::to_string(computeFramesTotal) + " Print Time: " + std::to_string(printFramesTotal) + "                      ";
        memcpy_s((void*)textFrame.data(), textFrame.size(), (void*)info.data(), info.size());

        auto printFrameStart = std::chrono::high_resolution_clock::now();
        SetConsoleCursorPosition(GetStdHandle(STD_OUTPUT_HANDLE), { 0, 0 });
        fwrite(textFrame.data(), textFrame.size(), 1, stdout);
        printFramesTotal = (std::chrono::high_resolution_clock::now() - printFrameStart).count() / 1000;
        
        // wait
        while ((std::chrono::high_resolution_clock::now() - startTime).count() / 1000 /* to MicroS */ < frameTime);
    }
    if (_cudaAccelEnabled)
    {
#ifdef __CUDA_RUNTIME_H__
        CudaFrameCompute::computeFrameFree();
#endif
    }
}

int main()
{
	// setup
	Renderer::setWidth(200);
	Renderer::allocBuffers();

	VideoPlayer::init();

	playVideo("./videos/Rick Astley - Never Gonna Give You Up.mp4", true);

	Renderer::freeBuffers();
    return 0;
}

