#include "hip/hip_runtime.h"
﻿//#include "cudaFunctions.h"

#include "VideoPlayer.h"
#include "frameCompute.h"


namespace CudaFrameCompute {
    __global__ void computeRow(uchar* _frame, char* _data, int* _width, size_t* _p0, char* _shadesDevice, int* _shadesDeviceLen)
    {
        int i = threadIdx.x;
        for (int j = 0; j < *_width; j++)
        {
            int offset = (*_p0) * i;

            unsigned __int8 v1 = _frame[*_width * 3 * i + j * 3];
            unsigned __int8 v2 = _frame[*_width * 3 * i + j * 3 + 1];
            unsigned __int8 v3 = _frame[*_width * 3 * i + j * 3 + 2];



            /*unsigned __int8* vec = &((_frame + (*_p0) * i))[j * 3];
            int offset = (*_p0) * i + j * 3;

            printf("v0%d\n", vec[0]);*/

            //printf("p%d\n", *_p0);
            //printf("f%d\n", _frame);
            //printf("v%d\n", vec);
            //printf("i%d\n", i);
            //printf("j%d\n", j);
            //printf("n%d\n", index);
            //printf("v%d\n", _frame[20]);
            //printf("t%d\n", _frame[index]);
            //printf("t%d\n", _frame[(*_p0) * i + j * 3]);

            //printf("v0%d\n", vec[0]);
            //printf("v0%d\n", vec[1]);
            //printf("v0%d\n", vec[2]);
            
            // calculate grayscale
            char color = (v1 + v2 + v3) / 3;
            // find fitting ASCII character
            if (color == 255)
            {
                color = _shadesDevice[*_shadesDeviceLen - 1];
            }
            else {
                color = _shadesDevice[(int)(color / (255.0 / *_shadesDeviceLen))];
            }
            // set ASCII character in final frame
            _data[((*_width) * 2 + 1) * i + (j + 1) * 2] = color;
            _data[((*_width) * 2 + 1) * i + (j + 1) * 2 - 1] = color;
        }
    }

    bool computeFrameAlloc(int _width, const cv::Mat& _frame, const std::string& _ASCIIFrame)
    {
        const char* shades = ".'`^\",:;Il!i><~+_-?][}{1)(|\\/tfjrxnuvczXYUJCLQ0OZmwqpdbkhao*#MW&8%B@";
        int shadesLen = strlen(shades);
        hipError_t cudaErr = hipSetDevice(0);
        if (cudaErr != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
            return false;
        }

        // alloc constant space
        hipMalloc(&CudaASCIIFrame, _ASCIIFrame.size());
        hipMalloc(&frame, _frame.dataend - _frame.datastart);
        
        // alloc constants
        hipMalloc(&width, sizeof(int));
        hipMalloc(&p0, sizeof(size_t));
        hipMalloc(&shadesDevice, strlen(shades));
        hipMalloc(&shadesDeviceLen, sizeof(int));
        
        // set constants
        hipMemcpy(width, &_width, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(p0, &_frame.size.p[0], sizeof(size_t), hipMemcpyHostToDevice);
        hipMemcpy(shadesDevice, shades, strlen(shades), hipMemcpyHostToDevice);
        hipMemcpy(shadesDeviceLen, &shadesLen, sizeof(int), hipMemcpyHostToDevice);
        return true;
    }

    bool computeFrameFree()
    {
        // free allocated 
        hipFree(width);
        hipFree(p0);
        hipFree(CudaASCIIFrame);
        hipFree(frame);
        hipFree(shadesDevice);
        hipFree(shadesDeviceLen);

        hipError_t cudeErr = hipDeviceReset();
        if (cudeErr != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
            return false;
        }
        return true;
    }


    bool computeFrameCuda(cv::Mat& _frame, std::string& _ASCIIFrame, int _width, int _height)
    {
        hipError_t cudaErr;
        // copy image data
        hipMemcpy(CudaASCIIFrame, _ASCIIFrame.data(), _ASCIIFrame.size(), hipMemcpyHostToDevice);
        // copy ACII frame
        hipMemcpy(frame, _frame.data, _frame.dataend - _frame.datastart, hipMemcpyHostToDevice);

        // launch kernel
        computeRow<<<1, _height>>>(frame, CudaASCIIFrame, width, p0, shadesDevice, shadesDeviceLen);

        cudaErr = hipGetLastError();
        if (cudaErr != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaErr));
            return false;
        }

        // wait for kernel to finish
        cudaErr = hipDeviceSynchronize();
        if (cudaErr != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaErr);
            return false;
        }

        hipMemcpy((void*)_ASCIIFrame.data(), CudaASCIIFrame, _ASCIIFrame.size(), hipMemcpyDeviceToHost);
        return true;
    }
}


void playVideo(std::string _filePath, bool _cudaAccelEnabled)
{
    cv::Mat frame;
    cv::Mat frameResized;

    cv::VideoCapture vidCap;
    // TODO: handle audio

    vidCap.open(1); // CAM
    //vidCap.open(_filePath, cv::CAP_FFMPEG);

    if (!vidCap.isOpened())
    {
        // todo: error msg
        return;
    }

    std::string textFrame;

    int height = 80;
    int width = vidCap.get(cv::CAP_PROP_FRAME_WIDTH) / vidCap.get(cv::CAP_PROP_FRAME_HEIGHT) * height;

    std::vector<int64_t> frameConvertTimes;
    //frameConvertTimes.reserve(vidCap.get(cv::CAP_PROP_FRAME_COUNT));

    textFrame.resize(height * (width * 2 + 1));
    // add new line to end of each row
    for (size_t i = 1; i < height; i++)
    {
        textFrame[(width * 2 + 1) * i] = '\n';
    }


    vidCap.get(cv::CAP_PROP_FRAME_COUNT);
    int frameTime = 1000 / vidCap.get(cv::CAP_PROP_FPS) * 1000;// frame time in MicroS
    bool cudeAllocated = false;
    for (auto startTime = std::chrono::high_resolution_clock::now();; startTime = std::chrono::high_resolution_clock::now()) {
        // read frame
        vidCap.read(frame);

        if (frame.empty())
            break;// video has ended

        // TODO: skip frames if to slow
                // resize frame
        cv::resize(frame, frameResized, cv::Size(width, height));// TODO: find best resize method

        // init cuda
        if (_cudaAccelEnabled && !cudeAllocated)
        {
#ifndef __CUDA_RUNTIME_H__
            std::cout << "No Cuda runtime detected!\n";
            return;
#else 
            if (!CudaFrameCompute::computeFrameAlloc(width, frameResized, textFrame))
            {
            	return;
            }
            cudeAllocated = true;
#endif
        }


        // create ASCII frame
        if (_cudaAccelEnabled)
        {// GPU
#ifdef __CUDA_RUNTIME_H__
            CudaFrameCompute::computeFrameCuda(frameResized, textFrame, width, height);
#endif // !__CUDA_RUNTIME_H__
        }
        else
        {// CPU
            VideoPlayer::computeFrame(frameResized, width, textFrame);
        }

        SetConsoleCursorPosition(GetStdHandle(STD_OUTPUT_HANDLE), { 0, 0 });
        fwrite(textFrame.data(), textFrame.size(), 1, stdout);
        // wait
        while ((std::chrono::high_resolution_clock::now() - startTime).count() / 1000 /* to MicroS */ < frameTime);
    }
    if (_cudaAccelEnabled)
    {
#ifdef __CUDA_RUNTIME_H__
        CudaFrameCompute::computeFrameFree();
#endif
    }
}

int main()
{
	// setup
	Renderer::setWidth(200);
	Renderer::allocBuffers();

	VideoPlayer::init();

	playVideo("./videos/badapple!.mp4", true);

	Renderer::freeBuffers();
    return 0;
}

